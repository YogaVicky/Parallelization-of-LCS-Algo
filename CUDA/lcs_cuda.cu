#include "hip/hip_runtime.h"
%%cu
#include<bits/stdc++.h>
#include<stdio.h>
using namespace std;
#include<chrono>
#define threads 4
__managed__ int dp[500][500];
__managed__ char X[500]={"AGGTABAGGTABAGGTABAGGTABAGGTABAGGTABAGGTABAGGTABAGGTABAGGTABGXTXAYB"};
__managed__ char Y[500]={"GXTXAYBGXTXAYBGXTXAYBGXTXAYBGXTXAYBGXTXAYBGXTXAYBGXTXAYBGXTXAYBXXXX"};


__global__ void test(int *d_a,int *d_b,int *d_c,int *d_d,int *d_e,int *d_f){

    int ix=threadIdx.x+blockIdx.x*blockDim.x;
    int ii=*d_d+*d_f;
    int jj=*d_e-*d_f;

    int val=0;
    if (ii == 0 || jj == 0)
        val = 0;
    else if (X[ii - 1] == Y[jj - 1])
        val = dp[ii - 1][jj - 1] + 1;
    else
        val = max(dp[ii - 1][jj], dp[ii][jj - 1]);
        d_a[ix]=ii;
        d_b[ix]=jj;
        d_c[ix]=val;

}


int lcs( string X, string Y, int m, int n ){
    int i,j,ii,jj,k,g,h;
    memset(dp,0,sizeof(dp));
    hipEvent_t start, stop;
		float elapsedTime;

    hipEventCreate(&start);
    hipEventRecord(start,0);

    for(int i=0,j = 0; j <= n && i <= m; j++){	
		int sz = min(j,(int)m-i);
    
    int size = (sz+1)*sizeof(int);
    int size1 = 1*sizeof(int);


    
    int *d_a,*d_b,*d_c,*d_d,*d_e,*d_f;

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    hipMalloc((void **)&d_d, size1);
    hipMalloc((void **)&d_e, size1);
    hipMalloc((void **)&d_f, size1);

    
    
            for(int k=0;k<=sz;++k){	
                	
                int i1[sz+1],i2[sz+1],i3[sz+1];

                hipMemcpy(d_d, &i, size1, hipMemcpyHostToDevice);
                hipMemcpy(d_e, &j, size1, hipMemcpyHostToDevice);
                hipMemcpy(d_f, &k, size1, hipMemcpyHostToDevice);

                test<<<(sz+1)/threads + 1,threads>>>(d_a,d_b,d_c,d_d,d_e,d_f);
                hipDeviceSynchronize();

                hipMemcpy(&i1, d_a, size, hipMemcpyDeviceToHost);
                hipMemcpy(&i2, d_b, size, hipMemcpyDeviceToHost);
                hipMemcpy(&i3, d_c, size, hipMemcpyDeviceToHost);

                for(int i=0;i<=sz;i++)
                {
                    dp[i1[i]][i2[i]]=i3[i];
                }
            }					
		if(j>=n)								
			j=n-1,i++;
	}	

  hipEventCreate(&stop);
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start,stop);
  printf("No of Threads=%d\nElapsed time (in seconds): %f\n" ,threads,elapsedTime/1000);

	return dp[m][n];
}

/* Driver program to test above function */
int main()
{
int m=67;
int n=67;

printf("Length of LCS is %d", lcs( X, Y, m, n ) );
hipDeviceSynchronize();
return 0;
}